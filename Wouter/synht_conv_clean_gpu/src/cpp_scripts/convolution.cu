#include "hip/hip_runtime.h"
#include <iostream>
#include "../headers/convolution.h"

Convolution::Convolution(std::vector<float> IRP){
  ImpulseResponse = IRP;
  SampleLength = IRP.size();
  Counter = 0;
  Convs.reserve(SampleLength + 1);
  InputVector.reserve(SampleLength + 1);
}

Convolution::~Convolution() {

}

__global__
void gpuCalc(std::vector<float> IR, std::vector<float> IN, int i) {
    int th = threadIdx.x + i; // the id number of our thread
    outputvar += IR[th] * IN[th];

}

float Convolution::getSamp(float CurSampy) {
  float Output = 0;
  outputvar = 0;
  //Makes a new Class instance for each sample that's put in.
  //That sample plays back the Impulse Response and multiplies it with the og sample it got.
  //Once it has played the full Impulse Response has played it gets removed.

  //Remove float from InputVector when it has played
  if(Counter >= SampleLength) {
    InputVector.pop_back();
  }
  //std::cout << "Kaas\n";
  InputVector.insert(InputVector.begin(), CurSampy);
  //InputVector.push_back(CurSampy);
  //Makes a new IR instance
  //Convs.push_back(SampleConv(ImpulseResponse, CurSampy));
  //Adds all Impulse Response values
  //  for(size_t i = 0; i < InputVector.size(); ++i) {
  //    Output += InputVector[i] * ImpulseResponse[i];
  //  }
  for(int i = 0; i < InputVector.size()) {
      if (InputVector.size() - i < 1024) {
          gpuCalc<<<1, InputVector.size() - i>>>(ImpulseResponse, InputVector, i);
          hipDeviceSynchronize();
          i += InputVector - i;
      } else {
          gpuCalc<<<1, 1024>>>(ImpulseResponse, InputVector, i);
          hipDeviceSynchronize();
          i += 1024;
      }
  }
  ++Counter;
  //std::cout << Output << "\n";
  //Output = Output / InputVector.size() * 50;
  //std::cout << Output << "\n";
  return outputvar;
}
